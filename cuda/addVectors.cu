#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#define N 10

__global__ void add (int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N) {
        c[tid] = a[tid]+b[tid];
    }
}
int main(void) {
    int a[N],b[N],c[N];
    // float a_h[N];
    int *dev_a, *dev_b, *dev_c;//, *a_d;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));
    
    // hipMalloc((void **) &a_d, sizeof(float)*N); // alokuj pamięć na GPU
    
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N*sizeof(int), hipMemcpyHostToDevice);
    // hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    
    add <<<1,N>>> (dev_a,dev_b,dev_c);
    
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d+%d=%d\n", a[i], b[i], c[i]);
    }
    
    // hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    printf ("Time for the kernel: %f ms\n", time);
    
    // hipFree(a_d);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
